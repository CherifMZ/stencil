
#include <hip/hip_runtime.h>
#include <stdio.h>

// Stencil kernel
__global__ void kernel(float *dA, float *dB, int N)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t j = blockIdx.y * blockDim.y + threadIdx.y;
    size_t k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < N && j < N && k < N) {
        dA[i * N + j * N + k] = 0.8 * (dB[(i - 1) * N + j * N + k] + dB[(i + 1) * N + N * j + k] + dB[i * N + (j - 1) * N + k] + 
                                dB[i * N + (j + 1) * N + k] + dB[i * N + j * N + (k - 1)] + dB[i * N + j * N  + (k + 1)]);
    }
}

//Performing the stencil using its kernel
extern "C" void stencil(float *hA, float *hB, int N)
{
    float *dA; 
    float *dB; 

    hipMalloc(&dA, sizeof(float) * N * N * N);
    hipMalloc(&dB, sizeof(float) * N * N * N);

    hipMemcpy(dA, hA, sizeof(float) * N * N * N, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(float) * N * N * N, hipMemcpyHostToDevice);

    dim3 dimBlock; 
    dimBlock.x = 32; 
    dimBlock.y = 32;
    dimBlock.z = 32; 

    dim3 dimGrid; 
    dimGrid.x = (N + dimBlock.x - 1) / dimBlock.x; 
    dimGrid.y = (N + dimBlock.y - 1) / dimBlock.y; 
    dimGrid.z = (N + dimBlock.z - 1) / dimBlock.z;

    kernel<<<dimGrid, dimBlock>>>(dA, dB, N);

    hipMemcpy(hA, dA, sizeof(float) * N * N * N, hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);

}